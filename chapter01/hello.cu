#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>

/*
 * A simple introduction to programming in CUDA. This program prints "Hello
 * World from GPU! from 100 CUDA threads running on the GPU.
 */

__global__ void helloFromGPU()
{
    if (threadIdx.x == 5)
    {
        printf("Hello World from GPU thread %i!\n", threadIdx.x);
    }
}

int main(int argc, char **argv)
{
    printf("Hello World from CPU!\n");

    helloFromGPU<<<1, 10>>>();
    CHECK(hipDeviceReset());
    return 0;
}