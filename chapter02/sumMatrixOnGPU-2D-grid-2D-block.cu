#include <hip/hip_runtime.h> 
#include <stdio.h> 
#include <sys/time.h>

#define CHECK(call)                      \
{                                        \
    const hipError_t error = call;      \
    if (error != hipSuccess)            \
    {                                    \
        printf("Error: %s:%d, ",         \
               __FILE__, __LINE__);      \
        printf("code:%d, reason: %s\n",  \
               error, hipGetErrorString(error)); \
        exit(1);                         \
    }                                    \
}

double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


int main(int argc, char **argv) { 
    printf("%s Starting...\n", argv[0]);
// set up device
int dev = 0;
hipDeviceProp_t deviceProp; 
CHECK(hipGetDeviceProperties(&deviceProp, dev)); 
printf("Using Device %d: %s\n", dev, deviceProp.name); 
CHECK(hipSetDevice(dev));
// set up date size of matrix
int nx = 1<<14;
int ny = 1<<14;
int nxy = nx*ny;
int nBytes = nxy * sizeof(float); 
printf("Matrix size: nx %d ny %d\n",nx, ny);
// malloc host memory
float *h_A, *h_B, *hostRef, *gpuRef; 
h_A = (float *)malloc(nBytes);
h_B = (float *)malloc(nBytes); 
hostRef = (float *)malloc(nBytes); 
gpuRef = (float *)malloc(nBytes);
// initialize data at host side
double iStart = cpuSecond(); 
initialData (h_A, nxy); 
initialData (h_B, nxy);
double iElaps = cpuSecond() - iStart;
memset(hostRef, 0, nBytes); 
memset(gpuRef, 0, nBytes);
// add matrix at host side for result checks
iStart = cpuSecond();
sumMatrixOnHost (h_A, h_B, hostRef, nx,ny); 
iElaps = cpuSecond() - iStart;
// malloc device global memory
float *d_MatA, *d_MatB, *d_MatC; 
hipMalloc((void **)&d_MatA, nBytes); 
hipMalloc((void **)&d_MatB, nBytes); 
hipMalloc((void **)&d_MatC, nBytes);
// transfer data from host to device
hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice); 
hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice);